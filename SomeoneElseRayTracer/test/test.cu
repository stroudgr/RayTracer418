#include <stdio.h>
#include <stdlib.h>
#include "kernel.h"
#include "util.h"
#include "shader.h"



int main(int argc, char *argv[]) {

    /*  Debug thing */
    vec3 *testVec;
    checkCudaErrors(hipMallocManaged((void **)&testVec, sizeof(vec3)));
    checkCudaErrors(hipDeviceSynchronize());

    int width   = 1920;
    int height  = 1080;
    /* Camera parameter   */
    vec3 pos    = vec3(0.0, 1.0, 6.0);
    vec3 look   = vec3(0.0, 0.0, 0.0);
    vec3 up     = vec3(0.0, 1.0, 0.0);

    /* Initialize camera  */
    Camera **camera;
    checkCudaErrors(hipMalloc((void **)&camera, sizeof(Camera *)));
    creatCamera<<<1,1>>>(camera, pos, look, up, width, height);
    checkCudaErrors(hipDeviceSynchronize());

    /* Initialize Light Sources */
    int N_light = 2;
    Light **lights;
    checkCudaErrors(hipMalloc((void **)&lights, sizeof(Light *) * N_light));
    createLight<<<1,1>>>(lights, 0, vec3(0,5,6), vec3(1,1,1), 200, testVec); // TODO: parse from file 
    createLight<<<1,1>>>(lights, 1, vec3(-4,2,6), vec3(1,1,1), 200, testVec); // TODO: parse from file 
    // createLight<<<1,1>>>(lights, 2, vec3(0,-3,6), vec3(0,1,0), 10, testVec); // TODO: parse from file 
    printf("Test Vec3 : %f, %f, %f\n", (*testVec)[0], (*testVec)[1], (*testVec)[2]);

    /* Initialize objects */
    int N_objs = 3; // number of objects
    Object **objs;
    checkCudaErrors(hipMalloc((void **)&objs, sizeof(Object *) * N_objs));
    addSphere<<<1,1>>>(objs); // TODO: make it flexibale
    checkCudaErrors(hipDeviceSynchronize());

    /* Initialize world */
    World **world;
    checkCudaErrors(hipMalloc((void **)&world, sizeof(World *)));
    createWorld<<<1,1>>>(world, camera, objs,N_objs, lights, N_light, vec3(0,0,0), 0, testVec );
    printf("Test Vec3 : %f, %f, %f\n", (*testVec)[0], (*testVec)[1], (*testVec)[2]);


    /* Initialize Shader */
    Shader **shader;
    checkCudaErrors(hipMalloc((void **)&shader, sizeof(Shader *)));
    createShader<<<1,1>>>(shader, vec3(1,0,0), vec3(1,0,0),vec3(1,1,1), 50, testVec );
    printf("Test Vec3 : %f, %f, %f\n", (*testVec)[0], (*testVec)[1], (*testVec)[2]);

    /* Initialize imgs */
    ivec3 *colors;
    checkCudaErrors(hipMallocManaged((void **)&colors, sizeof(ivec3)*height*width));
    checkCudaErrors(hipDeviceSynchronize());

    /* Render            */
    int tx = 8;
    int ty = 8;
    dim3 blocks(width/tx+1, height/ty +1);
    dim3 threads(tx, ty);
    render<<<blocks, threads>>>(shader, world, colors, width, height,testVec);
    checkCudaErrors(hipDeviceSynchronize());

    // Save img
    Dump_png(colors, width, height, "./result/test.png");
    
    /* Print debug info */
    printf("Color of first pixel: %d, %d, %d\n", colors[0][0], colors[0][1], colors[0][2]);
    printf("Test Vec3 : %f, %f, %f\n", (*testVec)[0], (*testVec)[1], (*testVec)[2]);


  return 0;
}
